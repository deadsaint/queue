#include "hip/hip_runtime.h"
//head always point to the flag one
//tail always point to last or last second one
//init:head = tail = flag one 
//malloc and free in enqueue && dequeue
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

using namespace std;

typedef struct node{
	//TODO any data type
	int data;
	struct node * next;
}node, * pnode;

typedef struct queue{
	pnode head;
	pnode tail;
}queue, *pqueue;

//__host__ void init(pqueue myqueue);

__device__ void enqueue(int mydata,pqueue myqueue);
__device__ int dequeue(pnode mynode, pqueue myqueue);
__device__ pnode myAtomicCAS(pnode * address, pnode compare, pnode val);
__device__ void deleteNode(pnode delnode);

//__global__ void app_bfs(pqueue myqueue, pnode d_dummy); // TODO add bfs to test queue.
__global__ void app_bfs(pqueue myqueue); // TODO add bfs to test queue.
__global__ void init(pqueue myqueue);
__global__ void show(pqueue myqueue);


int main(int argc, char * argv[]){
	int num_block, num_thread_perblock;
	
	//init and copy
	pnode h_dummy;
	pnode d_dummy;
	pqueue d_myqueue;

	if(argc != 3){
		printf("Usage: queue block_num thread_num\n");
		exit(1);
	}
	num_block = atoi(argv[1]);
	num_thread_perblock = atoi(argv[2]);

	h_dummy = (pnode)malloc(sizeof(node));
	h_dummy->data = -1;
	h_dummy->next = NULL;

	hipMalloc((void **)&d_dummy, sizeof(node));
	hipMalloc((void **)&d_myqueue, sizeof(queue));
	hipMemcpy(d_dummy, h_dummy, sizeof(node), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
 	float elapsedTime;
 	hipEventCreate(&start);
 	hipEventCreate(&stop);
 	hipEventRecord(start, 0);

	init<<<1,1>>>(d_myqueue);
	app_bfs<<<num_block,num_thread_perblock>>>(d_myqueue);
	show<<<1,1>>>(d_myqueue);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("[Info]Block:%d\tThread:%d\tElapsedTime:%fms\n", num_block, num_thread_perblock, elapsedTime); 

	hipEventDestroy(start);
	hipEventDestroy(stop);

	free(h_dummy);
	hipFree(d_dummy);
	hipFree(d_myqueue);
	//hipDeviceSynchronize();
	printf("[Info]%s\n",hipGetErrorString(hipGetLastError()));

	printf("[Info]Complete!\n");
	return 0;
}


__global__ void init(pqueue myqueue){
	pnode d_dummy = (pnode)malloc(sizeof(node));
	d_dummy->data = -1;
	d_dummy->next = NULL;
	myqueue->head = d_dummy;
	myqueue->tail = d_dummy;
}

__global__ void show(pqueue myqueue){
   pnode temp = myqueue->head;
   while(temp != NULL){
   	printf("%d\t",temp->data);
   	temp = temp->next;
   }
   printf("\n");
	
}

__global__ void app_bfs(pqueue myqueue){
	//printf("%d:%d\n", threadIdx.x, d_dummy->data);
/*
	if((blockIdx.x == 0) && ( threadIdx.x == 0)){
		pnode d_dummy = (pnode)malloc(sizeof(node));
		d_dummy->data = -1;
		d_dummy->next = NULL;
		myqueue->head = d_dummy;
		myqueue->tail = d_dummy;
	}
	__syncthreads();
*/
	//printf("[Info:start]block%d:thread:%d\n", blockIdx.x, threadIdx.x);
	pnode newnode = (pnode)malloc(sizeof(node));
	//enqueue(blockIdx.x * blockDim.x + threadIdx.x, myqueue);

	if(blockIdx.x % 2 == 1){
	//if(1){
		//printf("block:%d\tthread:%d\n", blockIdx.x, threadIdx.x);
		enqueue(blockIdx.x * blockDim.x + threadIdx.x, myqueue);
	}
	else{
		//printf("block:%d\tthread:%d\n", blockIdx.x, threadIdx.x);
		dequeue(newnode, myqueue);
		
//		if (!dequeue(newnode, myqueue))
//			printf("Block:%d Thread:%d out:%d\n", blockIdx.x, threadIdx.x, newnode->data);
//		else
//			printf("Block:%d Thread:%d out:NULL\n", blockIdx.x, threadIdx.x);
	    
	}

/*
	__syncthreads();
	if((blockIdx.x == 0) && ( threadIdx.x == 0)){
    	pnode temp = myqueue->head;
    	while(temp != NULL){
    		printf("%d\t",temp->data);
    		temp = temp->next;
    	}
		printf("\n");
	}
*/
	//printf("[Info:end]block%d:thread:%d\n", blockIdx.x, threadIdx.x);
}

__device__ pnode myAtomicCAS(pnode * address, pnode compare, pnode val){
	return (pnode)atomicCAS((unsigned long long int*)address,
			(unsigned long long int)compare,
			(unsigned long long int)val);
}
/*
//TODO in host
__host__ void init(pqueue myqueue){
	pnode mynode = new node();
	mynode->data = 0;
	mynode->next = NULL;
	myqueue->head = myqueue->head = mynode;
}
*/

//__device__ void enqueue(pnode newnode,pqueue myqueue){
__device__ void enqueue(int newdata,pqueue myqueue){
	int count = 0;
	pnode tail = NULL,next = NULL;
	pnode newnode = (pnode)malloc(sizeof(node));
/*
   if (newnode == NULL){// added can avoid the unspecified launch failure!!!
		printf("[Error]Malloc failed!\n");
		return ;
	}
*/
	newnode->data = newdata;
	newnode->next = NULL;
	//printf("In:enqueue:%d\n", threadIdx.x);
	
	while(1){
		tail = myqueue->tail;
		next = tail->next;
		if(tail == myqueue->tail){
			//printf("%dIn:tail==queue_tail\n",threadIdx.x);
			if(next == NULL){
				//printf("%dnext==NULL\n", threadIdx.x);
				if(next == myAtomicCAS(&myqueue->tail->next, next, newnode)){
					printf("Block:%d Thread:%d in:%d\n", blockIdx.x, threadIdx.x, newnode->data);
					break;
				}
			}
			else{ 
				//printf("%dnext != NULL\n",threadIdx.x);
				myAtomicCAS(&myqueue->tail, tail, next);// success or not both ok
			}
		}
	}
	myAtomicCAS(&myqueue->tail, tail, newnode); // success or not both ok
	
	//printf("Out:enqueue\n");
}

__device__ int dequeue(pnode mynode, pqueue myqueue){
	pnode tail = NULL;
	pnode head = NULL;
	pnode next = NULL;
	while(1){
		head = myqueue->head;
		tail = myqueue->tail;
		next = head->next;
		//printf("In:dequeue\n");
		if(head == myqueue->head){
			if(head == tail){
				//printf("In:head == tail\n");
				if(next == NULL){
					printf("Block:%d Thread:%d out:NULL\n", blockIdx.x, threadIdx.x);
					return -1;
				}
				else
					myAtomicCAS(&myqueue->tail, tail, next); // just try to do that...
			}
			else{
				//printf("In:head!=tail\n");
				mynode->data = next->data;
				if(head == myAtomicCAS(&myqueue->head, head, next)){
					//printf("out:%d\n",mynode->data);
					printf("Block:%d Thread:%d out:%d\n", blockIdx.x, threadIdx.x, mynode->data);
					break;
				}
			}
		}
	}

	//TODO first we dont delete node
	//deleteNode(head);

	return 0;
}

__device__ void deleteNode(pnode delnode){
	free(delnode);//TODO:delete node use memory reclamation
}
